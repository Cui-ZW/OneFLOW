#include "hip/hip_runtime.h"
/*---------------------------------------------------------------------------*\
    OneFLOW - LargeScale Multiphysics Scientific Simulation Environment
    Copyright (C) 2017-2021 He Xin and the OneFLOW contributors.
-------------------------------------------------------------------------------
License
    This file is part of OneFLOW.

    OneFLOW is free software: you can redistribute it and/or modify it
    under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    OneFLOW is distributed in the hope that it will be useful, but WITHOUT
    ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or
    FITNESS FOR A PARTICULAR PURPOSE.  See the GNU General Public License
    for more details.

    You should have received a copy of the GNU General Public License
    along with OneFLOW.  If not, see <http://www.gnu.org/licenses/>.

\*---------------------------------------------------------------------------*/

#include "AAASolverCuda.h"
#include "HXMath.h"
#include "Constant.h"
#ifdef ENABLE_CUDA
#include <hip/hip_runtime.h>
#endif
#include <iostream>

BeginNameSpace( ONEFLOW )

__global__ void SetValueKernel(Real *dev_a, Real *dev_b, int *dev_id );
__global__ void addKernel(int *a, int *b, int *c );

__global__ void SetValueKernel(Real *dev_a, Real *dev_b, int *dev_id )
{
    int iface = threadIdx.x;
    int icell = dev_id[ iface ];
    dev_a[ iface ] = dev_b[ icell ];
}

__global__ void SetValueKernelReal(Real *dev_a, Real *dev_b, int *dev_id, int nFaces, int nTCells )
{
    int iface = blockDim.x * blockIdx.x + threadIdx.x;
    if ( iface < nFaces )
    {
        int jcell = dev_id[ iface ];
        dev_a[iface] = dev_b[jcell];
    }
}

__global__ void MyInvFluxCuda(Real * qf1, Real * qf2, Real * invflux, Real * xfn, Real * yfn, Real * zfn, Real * area, int nFaces )
{
    int iFace = blockDim.x * blockIdx.x + threadIdx.x;
    if ( iFace < nFaces )
    {
        Real vxl = 1.0;
        Real vyl = 0.0;
        Real vzl = 0.0;

        Real vxr = 1.0;
        Real vyr = 0.0;
        Real vzr = 0.0;

        Real q_L = qf1[ iFace ];
        Real q_R = qf2[ iFace ];

        Real vnl  = xfn[ iFace ] * vxl + yfn[ iFace ] * vyl + zfn[ iFace ] * vzl;
        Real vnr  = xfn[ iFace ] * vxr + yfn[ iFace ] * vyr + zfn[ iFace ] * vzr;

        Real eigenL = vnl;
        Real eigenR = vnr;

        //eigenL = half * ( eigenL + ABS( eigenL ) );
        //eigenR = half * ( eigenR - ABS( eigenR ) );
        eigenL = 0.5 * ( eigenL + abs( eigenL ) );
        eigenR = 0.5 * ( eigenR - abs( eigenR ) );

        Real fL = q_L * eigenL;
        Real fR = q_R * eigenR;
        Real fM = fL + fR;

        Real areaM = area[ iFace ];
        invflux[ iFace ] = fM * areaM;
    }
}

__global__ void MyAddF2CFieldCudaDevice(Real * fField, Real * cField, int * lc, int * rc, int nBFaces, int nFaces )
{
    int iFace = blockDim.x * blockIdx.x + threadIdx.x;
    if ( iFace < nBFaces )
    {
        int lc_ = lc[ iFace ];
        Real value = fField[ iFace ];
        //cField[ lc_ ] -= fField[ iFace ];
        atomicAdd( &cField[ lc_ ], - value );
    }
    else if ( ( iFace >= nBFaces ) && ( iFace < nFaces ) )
    {
        int lc_ = lc[ iFace ];
        int rc_ = rc[ iFace ];

        Real value = fField[ iFace ];
        //cField[ lc_ ] -= fField[ iFace ];
        //cField[ rc_ ] += fField[ iFace ];
        atomicAdd( &cField[ lc_ ], - value );
        atomicAdd( &cField[ rc_ ], value );
    }
}

__global__ void MyAddF2CFieldCudaDeviceNoAtomic(Real * fField, Real * cField, int * lc, int * rc, int nBFaces, int nFaces )
{
    int iFace = blockDim.x * blockIdx.x + threadIdx.x;
    if ( iFace < nBFaces )
    {
        int lc_ = lc[ iFace ];
        cField[ lc_ ] -= fField[ iFace ];
    }
    else if ( ( iFace >= nBFaces ) && ( iFace < nFaces ) )
    {
        int lc_ = lc[ iFace ];
        int rc_ = rc[ iFace ];

        cField[ lc_ ] -= fField[ iFace ];
        cField[ rc_ ] += fField[ iFace ];
    }
}

__global__ void MyZoneTimeIntergralCudaDevice(Real * res, Real * vol, Real dt, int nCells)
{
    int iCell = blockDim.x * blockIdx.x + threadIdx.x;
    if ( iCell < nCells )
    {
        Real ovol = 1.0 / vol[ iCell ];
        Real coef = dt * ovol;
        res[ iCell ] *= coef;
    }
}

__global__ void MyZoneUpdateCudaDevice(Real *q, Real *res, int nCells)
{
    int iCell = blockDim.x * blockIdx.x + threadIdx.x;
    if ( iCell < nCells )
    {
        q[ iCell ] += res[ iCell ];
    }
}

void addWithCuda(int *a, int *b, int *c, unsigned int nElems)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;

    // Choose which GPU to run on, change this on a multi-GPU system.
    hipSetDevice(0);

    hipMalloc((void**)&dev_a, nElems * sizeof(int));
    hipMalloc((void**)&dev_b, nElems * sizeof(int));
    hipMalloc((void**)&dev_c, nElems * sizeof(int));

    hipMemcpy(dev_a, a, nElems * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, nElems * sizeof(int), hipMemcpyHostToDevice);

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, nElems>>>(dev_a, dev_b, dev_c);

    hipDeviceSynchronize();

    hipMemcpy(c, dev_c, nElems * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
}

__global__ void addKernel(int *a, int *b, int *c )
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

__global__ void addRealKernel(Real *a, Real *b, Real *c )
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

__global__ void addRealSwapKernel(Real *a, Real *b, int *id, Real *c )
{
    int i = threadIdx.x;
    int j = id[ i ];
    c[i] = a[i] + b[j];
}

__global__ void setRealSwapKernel(Real *a, int *id, Real *c )
{
    int i = threadIdx.x;
    int j = id[ i ];
    c[i] = a[j];
}

__global__ void setRealSwapKernelNew(Real *a, Real *b, int *id  )
{
    int i = threadIdx.x;
    int j = id[ i ];
    a[i] = b[j];
}

__global__ void setRealSwapKernelNew1(Real *a, Real *b, int *id, int nElems )
{
    //int i = threadIdx.x;
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if ( i < nElems )
    {
        int j = id[ i ];
        a[i] = b[j];
    }
}

void addRealWithCuda(Real *a, Real *b, Real *c, unsigned int nElems)
{
    Real *dev_a = 0;
    Real *dev_b = 0;
    Real *dev_c = 0;

    // Choose which GPU to run on, change this on a multi-GPU system.
    hipSetDevice(0);

    hipMalloc((void**)&dev_a, nElems * sizeof(Real));
    hipMalloc((void**)&dev_b, nElems * sizeof(Real));
    hipMalloc((void**)&dev_c, nElems * sizeof(Real));

    hipMemcpy(dev_a, a, nElems * sizeof(Real), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, nElems * sizeof(Real), hipMemcpyHostToDevice);

    // Launch a kernel on the GPU with one thread for each element.
    addRealKernel<<<1, nElems>>>(dev_a, dev_b, dev_c);

    hipDeviceSynchronize();

    hipMemcpy(c, dev_c, nElems * sizeof(Real), hipMemcpyDeviceToHost);

    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
}

void addRealSwapWithCuda(Real *a, Real *b, int * id, Real *c, unsigned int nElems)
{
    Real *dev_a = 0;
    Real *dev_b = 0;
    Real *dev_c = 0;
    int * dev_id = 0;

    // Choose which GPU to run on, change this on a multi-GPU system.
    hipSetDevice(0);

    hipMalloc((void**)&dev_a, nElems * sizeof(Real));
    hipMalloc((void**)&dev_b, nElems * sizeof(Real));
    hipMalloc((void**)&dev_c, nElems * sizeof(Real));
    hipMalloc((void**)&dev_id, nElems * sizeof(Real));

    hipMemcpy(dev_a, a, nElems * sizeof(Real), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, nElems * sizeof(Real), hipMemcpyHostToDevice);
    hipMemcpy(dev_id, id, nElems * sizeof(int), hipMemcpyHostToDevice);

    // Launch a kernel on the GPU with one thread for each element.
    addRealSwapKernel<<<1, nElems>>>(dev_a, dev_b, dev_id, dev_c);

    hipDeviceSynchronize();

    hipMemcpy(c, dev_c, nElems * sizeof(Real), hipMemcpyDeviceToHost);

    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_id);
}

void setRealSwapWithCuda(Real *a, int * id, Real *c, unsigned int nElems)
{
    Real *dev_a = 0;
    Real *dev_c = 0;
    int * dev_id = 0;

    // Choose which GPU to run on, change this on a multi-GPU system.
    hipSetDevice(0);

    hipMalloc((void**)&dev_a, nElems * sizeof(Real));
    hipMalloc((void**)&dev_c, nElems * sizeof(Real));
    hipMalloc((void**)&dev_id, nElems * sizeof(Real));

    hipMemcpy(dev_a, a, nElems * sizeof(Real), hipMemcpyHostToDevice);
    hipMemcpy(dev_id, id, nElems * sizeof(int), hipMemcpyHostToDevice);

    // Launch a kernel on the GPU with one thread for each element.
    setRealSwapKernel<<<1, nElems>>>(dev_a, dev_id, dev_c);

    hipDeviceSynchronize();

    hipMemcpy(c, dev_c, nElems * sizeof(Real), hipMemcpyDeviceToHost);

    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_id);
}

void setRealSwapWithCudaNew(Real *a, Real *b, int * id,  unsigned int nElems)
{
    Real *dev_a = 0;
    Real *dev_b = 0;
    int * dev_id = 0;

    // Choose which GPU to run on, change this on a multi-GPU system.
    hipSetDevice(0);

    hipMalloc((void**)&dev_a, nElems * sizeof(Real));
    hipMalloc((void**)&dev_b, nElems * sizeof(Real));
    hipMalloc((void**)&dev_id, nElems * sizeof(Real));

    hipMemcpy(dev_b, b, nElems * sizeof(Real), hipMemcpyHostToDevice);
    hipMemcpy(dev_id, id, nElems * sizeof(int), hipMemcpyHostToDevice);

    //setRealSwapKernelNew<<<1, nElems>>>(dev_a, dev_b, dev_id);

    int threadsPerBlock = 256;
    int blocksPerGrid =(nElems + threadsPerBlock - 1) / threadsPerBlock;

    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);

    setRealSwapKernelNew1<<<blocksPerGrid, threadsPerBlock>>>(dev_a, dev_b, dev_id, nElems);

    hipDeviceSynchronize();

    hipMemcpy(a, dev_a, nElems * sizeof(Real), hipMemcpyDeviceToHost);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_id);
}

void setRealSwapWithCudaNewRealProblem(Real *a, Real *b, int * id, unsigned int nFaces, unsigned int nCells)
{
    Real *dev_a = 0;
    Real *dev_b = 0;
    int * dev_id = 0;

    // Choose which GPU to run on, change this on a multi-GPU system.
    hipSetDevice(0);

    hipMalloc((void**)&dev_a, nFaces * sizeof(Real));
    hipMalloc((void**)&dev_b, nCells * sizeof(Real));
    hipMalloc((void**)&dev_id, nFaces * sizeof(Real));

    hipMemcpy(dev_b, b, nCells * sizeof(Real), hipMemcpyHostToDevice);
    hipMemcpy(dev_id, id, nFaces * sizeof(int), hipMemcpyHostToDevice);

    setRealSwapKernelNew<<<1, nFaces>>>(dev_a, dev_b, dev_id);

    hipDeviceSynchronize();

    hipMemcpy(a, dev_a, nFaces * sizeof(Real), hipMemcpyDeviceToHost);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_id);
}

void SetValueWithCuda(Real *aface, Real *bcell, int *id, unsigned int nFaces, unsigned int nTCells)
{
    Real *dev_a = 0;
    Real *dev_b = 0;
    int *dev_id = 0;

    hipSetDevice(0);

    hipMalloc((void**)&dev_a, nFaces * sizeof(Real));
    hipMalloc((void**)&dev_b, nTCells * sizeof(Real));
    hipMalloc((void**)&dev_id, nFaces * sizeof(int));

    hipMemcpy(dev_a, aface, nFaces * sizeof(Real), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, bcell, nTCells * sizeof(Real), hipMemcpyHostToDevice);
    hipMemcpy(dev_id, id, nFaces * sizeof(int), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (nFaces + threadsPerBlock - 1) / threadsPerBlock;

    //printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);

    SetValueKernelReal<<<blocksPerGrid, threadsPerBlock>>>(dev_a, dev_b, dev_id, nFaces, nTCells );

    hipDeviceSynchronize();

    hipMemcpy(aface, dev_a, nFaces * sizeof(Real), hipMemcpyDeviceToHost);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_id);
}

void MyCalcInvFluxCuda(Real *qf1, Real *qf2, Real *invflux, Real *xfn, Real *yfn, Real *zfn, Real *area, int nFaces)
{
    Real *dev_qf1 = 0;
    Real *dev_qf2 = 0;
    Real *dev_invflux = 0;
    Real *dev_xfn = 0;
    Real *dev_yfn = 0;
    Real *dev_zfn = 0;
    Real *dev_area = 0;

    hipSetDevice(0);

    hipMalloc((void**)&dev_qf1, nFaces * sizeof(Real));
    hipMalloc((void**)&dev_qf2, nFaces * sizeof(Real));
    hipMalloc((void**)&dev_invflux, nFaces * sizeof(Real));
    hipMalloc((void**)&dev_xfn, nFaces * sizeof(Real));
    hipMalloc((void**)&dev_yfn, nFaces * sizeof(Real));
    hipMalloc((void**)&dev_zfn, nFaces * sizeof(Real));
    hipMalloc((void**)&dev_area, nFaces * sizeof(Real));

    hipMemcpy(dev_qf1, qf1, nFaces * sizeof(Real), hipMemcpyHostToDevice);
    hipMemcpy(dev_qf2, qf2, nFaces * sizeof(Real), hipMemcpyHostToDevice);
    hipMemcpy(dev_invflux, invflux, nFaces * sizeof(Real), hipMemcpyHostToDevice);
    hipMemcpy(dev_xfn, xfn, nFaces * sizeof(Real), hipMemcpyHostToDevice);
    hipMemcpy(dev_yfn, yfn, nFaces * sizeof(Real), hipMemcpyHostToDevice);
    hipMemcpy(dev_zfn, zfn, nFaces * sizeof(Real), hipMemcpyHostToDevice);
    hipMemcpy(dev_area, area, nFaces * sizeof(Real), hipMemcpyHostToDevice);


    int threadsPerBlock = 256;
    int blocksPerGrid = (nFaces + threadsPerBlock - 1) / threadsPerBlock;

    //printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);

    MyInvFluxCuda<<<blocksPerGrid, threadsPerBlock>>>(dev_qf1, dev_qf2, dev_invflux, dev_xfn, dev_yfn, dev_zfn, dev_area, nFaces );

    hipDeviceSynchronize();

    hipMemcpy(invflux, dev_invflux, nFaces * sizeof(Real), hipMemcpyDeviceToHost);

    hipFree(dev_qf1);
    hipFree(dev_qf2);
    hipFree(dev_invflux);
    hipFree(dev_xfn);
    hipFree(dev_yfn);
    hipFree(dev_zfn);
    hipFree(dev_area);
}

void MyAddF2CFieldCuda(Real *fField, Real *cField, int *lc, int * rc, int nBFaces, int nFaces, int nTCells)
{
    Real *dev_fField = 0;
    Real *dev_cField = 0;
    int *dev_lc = 0;
    int *dev_rc = 0;

    hipSetDevice(0);

    hipMalloc((void**)&dev_fField, nFaces * sizeof(Real));
    hipMalloc((void**)&dev_cField, nTCells * sizeof(Real));
    hipMalloc((void**)&dev_lc, nFaces * sizeof(int));
    hipMalloc((void**)&dev_rc, nFaces * sizeof(int));

    hipMemcpy(dev_fField, fField, nFaces * sizeof(Real), hipMemcpyHostToDevice);
    hipMemcpy(dev_cField, cField, nTCells * sizeof(Real), hipMemcpyHostToDevice);
    hipMemcpy(dev_lc, lc, nFaces * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_rc, rc, nFaces * sizeof(int), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (nFaces + threadsPerBlock - 1) / threadsPerBlock;

    MyAddF2CFieldCudaDevice<<<blocksPerGrid, threadsPerBlock>>>(dev_fField, dev_cField, dev_lc, dev_rc, nBFaces, nFaces );
    //MyAddF2CFieldCudaDeviceNoAtomic<<<blocksPerGrid, threadsPerBlock>>>(dev_fField, dev_cField, dev_lc, dev_rc, nBFaces, nFaces );

    hipDeviceSynchronize();

    hipMemcpy(cField, dev_cField, nTCells * sizeof(Real), hipMemcpyDeviceToHost);

    hipFree(dev_fField);
    hipFree(dev_cField);
    hipFree(dev_lc);
    hipFree(dev_rc);
}


void MyZoneTimeIntergralCuda(Real *res, Real *vol, Real dt, int nCells)
{
    Real *dev_res = 0;
    Real *dev_vol = 0;

    hipSetDevice(0);

    hipMalloc((void**)&dev_res, nCells * sizeof(Real));
    hipMalloc((void**)&dev_vol, nCells * sizeof(Real));

    hipMemcpy(dev_res, res, nCells * sizeof(Real), hipMemcpyHostToDevice);
    hipMemcpy(dev_vol, vol, nCells * sizeof(Real), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (nCells + threadsPerBlock - 1) / threadsPerBlock;

    MyZoneTimeIntergralCudaDevice<<<blocksPerGrid, threadsPerBlock>>>(dev_res, dev_vol, dt, nCells);

    hipDeviceSynchronize();

    hipMemcpy(res, dev_res, nCells * sizeof(Real), hipMemcpyDeviceToHost);

    hipFree(dev_res);
    hipFree(dev_vol);

}

void MyZoneUpdateCuda(Real *q, Real *res, int nCells)
{
    Real *dev_res = 0;
    Real *dev_q = 0;

    hipSetDevice(0);

    hipMalloc((void**)&dev_res, nCells * sizeof(Real));
    hipMalloc((void**)&dev_q, nCells * sizeof(Real));

    hipMemcpy(dev_res, res, nCells * sizeof(Real), hipMemcpyHostToDevice);
    hipMemcpy(dev_q, q, nCells * sizeof(Real), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (nCells + threadsPerBlock - 1) / threadsPerBlock;

    MyZoneUpdateCudaDevice<<<blocksPerGrid, threadsPerBlock>>>(dev_q, dev_res, nCells);

    hipDeviceSynchronize();

    hipMemcpy(q, dev_q, nCells * sizeof(Real), hipMemcpyDeviceToHost);

    hipFree(dev_q);
    hipFree(dev_res);
}

EndNameSpace
