
#include <hip/hip_runtime.h>
#ifdef ENABLE_CUDA

#include <cuda_runtime.h>
#include <iostream>
using namespace std;

void printDeviceProp(const cudaDeviceProp& prop)
{
    std::cout << "Device Name : " << prop.name << "\n";
    std::cout << "totalGlobalMem : " << prop.totalGlobalMem << "\n";
    std::cout << "sharedMemPerBlock " << prop.sharedMemPerBlock << "\n";
    std::cout << "regsPerBlock : " << prop.regsPerBlock << "\n";
    std::cout << "warpSize :" << prop.warpSize << "\n";
    std::cout << "memPitch : " << prop.memPitch << "\n";
    std::cout << "maxThreadsPerBlock " << prop.maxThreadsPerBlock << "\n";
    std::cout << "maxThreadsDim[0 - 2] : " << prop.maxThreadsDim[0] << " " << prop.maxThreadsDim[1] << " " << prop.maxThreadsDim[2] << "\n";
    std::cout << "maxGridSize[0 - 2] " << prop.maxGridSize[0] << " " << prop.maxGridSize[1] << " " << prop.maxGridSize[2] << "\n";
    std::cout << "totalConstMem : " << prop.totalConstMem << "\n";
    std::cout << "major.minor : " << prop.major << "." << prop.minor << "\n";
    std::cout << "clockRate : " << prop.clockRate << "\n";
    std::cout << "textureAlignment :" << prop.textureAlignment << "\n";
    std::cout << "deviceOverlap : " << prop.deviceOverlap << "\n";
    std::cout << "multiProcessorCount : " << prop.multiProcessorCount << "\n";
}

bool InitCUDA()
{
    //used to count the device numbers
    int count;

    cudaGetDeviceCount(&count);
    if (count == 0) {
        fprintf(stderr, "There is no device.\n");
        return false;
    }

    // find the device >= 1.X
    int i;
    for (i = 0; i < count; ++i) {
        cudaDeviceProp prop;
        if (cudaGetDeviceProperties(&prop, i) == cudaSuccess) {
            if (prop.major >= 1) {
                printDeviceProp(prop);
                break;
            }
        }
    }

    // if can't find the device
    if (i == count) {
        fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
        return false;
    }

    // set cuda device
    cudaSetDevice(i);

    return true;
}

#endif // ENABLE_CUDA
